#include <stddef.h>
#include <vector>
#include <string>

#if defined(GPU_INFO_USE_HIPBLAS)
#include <hip/hip_runtime.h>
#include <hipblas/hipblas.h>
#define cudaGetDevice hipGetDevice
#define cudaGetDeviceCount hipGetDeviceCount
#define cudaGetErrorString hipGetErrorString
#define cudaMemGetInfo hipMemGetInfo
#define cudaSetDevice hipSetDevice
#define cudaSuccess hipSuccess
#else
#include <hip/hip_runtime.h>

#endif


typedef void (*gpuInfoCudaErrorLogCallback_t)(const char* message);

bool gpuInfoSetCudaDevice(const int device, gpuInfoCudaErrorLogCallback_t errorLogCallback) {
    int current_device;
    auto getDeviceResult = hipGetDevice(&current_device);

    if (getDeviceResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getDeviceResult));
        return false;
    }

    if (device == current_device) {
        return true;
    }

    const auto setDeviceResult = hipSetDevice(device);

    if (setDeviceResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(setDeviceResult));
        return false;
    }

    return true;
}

bool gpuInfoGetCudaDeviceInfo(int device, size_t * total, size_t * used, gpuInfoCudaErrorLogCallback_t errorLogCallback) {
    gpuInfoSetCudaDevice(device, errorLogCallback);

    size_t freeMem;
    size_t totalMem;
    auto getMemInfoResult = hipMemGetInfo(&freeMem, &totalMem);

    if (getMemInfoResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getMemInfoResult));
        return false;
    }

    *total = totalMem;
    *used = totalMem - freeMem;

    return true;
}

int gpuInfoGetCudaDeviceCount(gpuInfoCudaErrorLogCallback_t errorLogCallback) {
    int deviceCount;
    auto getDeviceCountResult = hipGetDeviceCount(&deviceCount);

    if (getDeviceCountResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getDeviceCountResult));
        return -1;
    }

    return deviceCount;
}

bool gpuInfoGetTotalCudaDevicesInfo(size_t * total, size_t * used, gpuInfoCudaErrorLogCallback_t errorLogCallback) {
    int deviceCount = gpuInfoGetCudaDeviceCount(errorLogCallback);

    if (deviceCount < 0) {
        return false;
    }

    size_t usedMem = 0;
    size_t totalMem = 0;

    for (int i = 0; i < deviceCount; i++) {
        size_t deviceUsedMem;
        size_t deviceTotalMem;

        if (!gpuInfoGetCudaDeviceInfo(i, &deviceTotalMem, &deviceUsedMem, errorLogCallback)) {
            return false;
        }

        usedMem += deviceUsedMem;
        totalMem += deviceTotalMem;
    }

    *total = totalMem;
    *used = usedMem;

    return true;
}

void gpuInfoGetCudaDeviceNames(std::vector<std::string> * deviceNames, gpuInfoCudaErrorLogCallback_t errorLogCallback) {
    int deviceCount = gpuInfoGetCudaDeviceCount(errorLogCallback);

    if (deviceCount < 0) {
        return;
    }

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        auto getDevicePropertiesResult = hipGetDeviceProperties(&prop, i);

        if (getDevicePropertiesResult != hipSuccess) {
            errorLogCallback(hipGetErrorString(getDevicePropertiesResult));
        } else {
            (*deviceNames).push_back(std::string(prop.name));
        }
    }
}
